#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cuh"
#include <stdio.h>

ptychofft::ptychofft(size_t Ntheta_, size_t Nz_, size_t N_, 
	size_t Nscanx_, size_t Nscany_, size_t detx_, size_t dety_, size_t Nprb_)
{
	N = N_;
	Ntheta = Ntheta_;
	Nz = Nz_;
	Nscanx = Nscanx_;
	Nscany = Nscany_;
	detx = detx_;
	dety = dety_;
	Nprb = Nprb_;

	hipMalloc((void**)&f,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2));
	hipMalloc((void**)&scanx,Ntheta*Nscanx*sizeof(int));
	hipMalloc((void**)&scany,Ntheta*Nscany*sizeof(int));
	hipMalloc((void**)&prb,Nprb*Nprb*sizeof(float2));
	hipMalloc((void**)&ff,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&fff,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&data,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float));
	hipMalloc((void**)&ftmp0,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&ftmp1,Ntheta*Nz*N*sizeof(float2));

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	ffts[0] = detx; ffts[1] = dety;
	idist = detx*dety; odist = detx*dety;
	inembed[0] = detx; inembed[1] = dety;
	onembed[0] = detx; onembed[1] = dety;
	hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nscanx*Nscany); 
}

ptychofft::~ptychofft()
{	
	hipFree(f);
	hipFree(g);
	hipFree(scanx);
	hipFree(scany);
	hipFree(prb);
	hipFree(ff);
	hipFree(fff);
	hipFree(data);
	hipFree(ftmp0);
	hipFree(ftmp1);
	hipfftDestroy(plan2dfwd);
}

void ptychofft::setobjc(int* scanx_, int* scany_, float2* prb_)
{
	hipMemcpy(scanx,scanx_,Ntheta*Nscanx*sizeof(int),hipMemcpyDefault);  	
	hipMemcpy(scany,scany_,Ntheta*Nscany*sizeof(int),hipMemcpyDefault);  	
	hipMemcpy(prb,prb_,Nprb*Nprb*sizeof(float2),hipMemcpyDefault);
}

void ptychofft::fwdc(float2* g_, float2* f_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(f,f_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	hipMemset(g,0,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2));

	mul<<<GS3d0,BS3d>>>(g,f,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);
	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);

	hipMemcpy(g_,g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::adjc(float2* f_, float2* g_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(g,g_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
	hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));

	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
	mula<<<GS3d0,BS3d>>>(f,g,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);

	hipMemcpy(f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::adjfwd_prbc(float2* f_, float2* ff_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));

	hipMemcpy(ff,ff_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	mulamul<<<GS3d0,BS3d>>>(f,ff,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);

	hipMemcpy(f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::update_ampc(float2* g_, float* data_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(detx*dety/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(g,g_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);
	hipMemcpy(data,data_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float),hipMemcpyDefault);
	updateamp<<<GS3d0,BS3d>>>(g,data,Ntheta,Nscanx*Nscany,detx*dety);
	hipMemcpy(g_,g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::grad_ptychoc(float2* f_,float* data_, float2* ff_, float2* fff_, float rho, 
	float gamma, float maxint, int niter)
{
	hipMemcpy(f,f_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	hipMemcpy(data,data_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float),hipMemcpyDefault);
	hipMemcpy(ff,ff_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	hipMemcpy(fff,fff_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);

	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));
	dim3 GS3d1(ceil(detx*dety/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	//fprintf(stderr,"%f %f %f %d\n",rho,gamma,maxint,niter);
	for (int i=0;i<niter;i++)
	{
		hipMemset(g,0,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2));
		hipMemset(ftmp0,0,Ntheta*Nz*N*sizeof(float2));
		hipMemset(ftmp1,0,Ntheta*Nz*N*sizeof(float2));

		mul<<<GS3d0,BS3d>>>(g,f,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);
		hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);
		updateamp<<<GS3d1,BS3d>>>(g,data,Ntheta,Nscanx*Nscany,detx*dety);
		hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
		mula<<<GS3d0,BS3d>>>(ftmp0,g,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);
		mulamul<<<GS3d0,BS3d>>>(ftmp1,f,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);
		updatepsi<<<GS3d0,BS3d>>>(f,ff,ftmp0,ftmp1,fff,rho,gamma,maxint,Ntheta,Nz,N);
	}
	hipMemcpy(f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);

}





void ptychofft::setobj(int* scanx_, int N30, int N31,
					int* scany_, int N40, int n41,
					float2* prb_, int N50, int N51)
{
	setobjc(scanx_, scany_, prb_);
}

void ptychofft::fwd(float2* g_, int N00, int N01, int N02, int N03,
					float2* f_, int N10, int N11, int N12)	
{
	fwdc(g_, f_);
}

void ptychofft::adj(float2* f_, int N10, int N11, int N12,
					float2* g_, int N00, int N01, int N02, int N03)	
{
	adjc(f_, g_);
}

void ptychofft::adjfwd_prb(float2* f_, int N10, int N11, int N12, float2* ff_, int N60, int N61, int N62)
{
	adjfwd_prbc(f_,ff_);
}


void ptychofft::update_amp(float2* g_, int N00, int N01, int N02, int N03,
	float* data_, int N70, int N71, int N72, int N73)
{
	update_ampc(g_,data_);
}

void ptychofft::grad_ptycho(
	float2* f_, int N10, int N11, int N12,
	float* data_, int N70, int N71, int N72, int N73,
	float2* ff_, int N60, int N61, int N62,
	float2* fff_, int N80, int N81, int N82,
	float rho, float gamma, float maxint, int niter)	
{
	grad_ptychoc(f_,data_,ff_,fff_,rho,gamma,maxint,niter);
}






