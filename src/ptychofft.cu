#include "hip/hip_runtime.h"
#include "ptychofft.cuh"
#include "kernels.cuh"
#include <stdio.h>

ptychofft::ptychofft(size_t Ntheta_, size_t Nz_, size_t N_, 
	size_t Nscanx_, size_t Nscany_, size_t detx_, size_t dety_, size_t Nprb_)
{
	N = N_;
	Ntheta = Ntheta_;
	Nz = Nz_;
	Nscanx = Nscanx_;
	Nscany = Nscany_;
	detx = detx_;
	dety = dety_;
	Nprb = Nprb_;

	hipMalloc((void**)&f,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2));
	hipMalloc((void**)&scanx,Ntheta*Nscanx*sizeof(int));
	hipMalloc((void**)&scany,Ntheta*Nscany*sizeof(int));
	hipMalloc((void**)&prb,Nprb*Nprb*sizeof(float2));
	hipMalloc((void**)&ff,Ntheta*Nz*N*sizeof(float2));
	hipMalloc((void**)&data,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float));

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	ffts[0] = detx; ffts[1] = dety;
	idist = detx*dety; odist = detx*dety;
	inembed[0] = detx; inembed[1] = dety;
	onembed[0] = detx; onembed[1] = dety;
	hipfftPlanMany(&plan2dfwd, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nscanx*Nscany); 
	fprintf(stderr,"ptycho created %d angles\n",Ntheta);
}

ptychofft::~ptychofft()
{	
	hipFree(f);
	hipFree(g);
	hipFree(scanx);
	hipFree(scany);
	hipFree(prb);
	hipFree(ff);
	hipFree(data);
	hipfftDestroy(plan2dfwd);
	fprintf(stderr,"ptycho removed\n");
}

void ptychofft::setobjc(int* scanx_, int* scany_, float2* prb_)
{
	hipMemcpy(scanx,scanx_,Ntheta*Nscanx*sizeof(int),hipMemcpyDefault);  	
	hipMemcpy(scany,scany_,Ntheta*Nscany*sizeof(int),hipMemcpyDefault);  	
	hipMemcpy(prb,prb_,Nprb*Nprb*sizeof(float2),hipMemcpyDefault);
}

void ptychofft::fwdc(float2* g_, float2* f_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(f,f_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	hipMemset(g,0,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2));

	mul<<<GS3d0,BS3d>>>(g,f,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);
	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);

	hipMemcpy(g_,g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::adjc(float2* f_, float2* g_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(g,g_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
	hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));

	hipfftExecC2C(plan2dfwd, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
	mula<<<GS3d0,BS3d>>>(f,g,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);

	hipMemcpy(f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::adjfwd_prbc(float2* f_, float2* ff_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(Nprb*Nprb/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemset(f,0,Ntheta*Nz*N*sizeof(float2));

	hipMemcpy(ff,ff_,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);
	mulamul<<<GS3d0,BS3d>>>(f,ff,prb,scanx,scany,Ntheta,Nz,N,Nscanx,Nscany,Nprb,detx,dety);

	hipMemcpy(f_,f,Ntheta*Nz*N*sizeof(float2),hipMemcpyDefault);  	
}

void ptychofft::update_ampc(float2* g_, float* data_)
{
	dim3 BS3d(32,32,1);
	dim3 GS3d0(ceil(detx*dety/(float)BS3d.x),ceil(Nscanx*Nscany/(float)BS3d.y),ceil(Ntheta/(float)BS3d.z));

	hipMemcpy(g,g_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);
	hipMemcpy(data,data_,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float),hipMemcpyDefault);
	updateamp<<<GS3d0,BS3d>>>(g,data,Ntheta,Nscanx*Nscany,detx*dety);
	hipMemcpy(g_,g,Ntheta*Nscanx*Nscany*detx*dety*sizeof(float2),hipMemcpyDefault);  	
}






void ptychofft::setobj(int* scanx_, int N30, int N31,
					int* scany_, int N40, int n41,
					float* prb_, int N50, int N51)
{
	setobjc(scanx_, scany_, (float2*)prb_);
}

void ptychofft::fwd(float* g_, int N00, int N01, int N02, int N03,
					float* f_, int N10, int N11, int N12)	
{
	fwdc((float2*)g_, (float2*)f_);
}

void ptychofft::adj(float* f_, int N10, int N11, int N12,
					float* g_, int N00, int N01, int N02, int N03)	
{
	adjc((float2*)f_, (float2*)g_);
}

void ptychofft::adjfwd_prb(float* f_, int N10, int N11, int N12, float* ff_, int N60, int N61, int N62)
{
	adjfwd_prbc((float2*)f_,(float2*)ff_);
}


void ptychofft::update_amp(float* g_, int N00, int N01, int N02, int N03,
	float* data_, int N70, int N71, int N72, int N73)
{
	update_ampc((float2*)g_,data_);
}








